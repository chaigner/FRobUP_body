
#include <hip/hip_runtime.h>
/* This is code by Zhipeng Cao and Will Grissom who have given permission 
% for inclusion within this package. This code can be found also in their
% repo https://bitbucket.org/wgrissom/acptx/ Please cite appropriately.*/
        
#define PI 3.14159265

__global__
void kGradHess( double *vx,
		double *Sx,
		double *vy,
		double *Sy,
		double *vz,
		double *Sz,
		double *Ar,
		double *Ai,
		double *dr,
		double *di,
		double *x,
		double *y,
		double *z,
		int *m2v,
		const int ns,
		const int nd,
		const int nr)
{

  int j = blockIdx.x*blockDim.x + threadIdx.x;
  double ajdr, ajdi, sjl, tmp;
  double ajai_i, ajai_r, sji, tmp2;  
  int i,k;
    
  /* separable term */
  while(j < ns){
    for(i=0;i<nr;i++){
	
      ajdr = dr[j]*Ar[i*ns+j] + di[j]*Ai[i*ns+j];
      ajdi = dr[j]*Ai[i*ns+j] - di[j]*Ar[i*ns+j];
      sjl = atan2(ajdi,ajdr);
      
      switch(nd){
      case 1:
	tmp = 2*PI*ajdi*x[j];
	vx[i*ns+j] = -tmp;
	if(sjl != 0)
	  Sx[m2v[i*nr+i]*ns+j] = 2*PI*x[j]*tmp/sjl;
	break;
      case 2:
	tmp = 2*PI*ajdi*x[j];
	vx[i*ns+j] = -tmp;
	if(sjl != 0)
	  Sx[m2v[i*nr+i]*ns+j] = 2*PI*x[j]*tmp/sjl;
	
	tmp = 2*PI*ajdi*y[j];
	vy[i*ns+j] = -tmp;
	if(sjl != 0)
	  Sy[m2v[i*nr+i]*ns+j] = 2*PI*y[j]*tmp/sjl;
	break;
      case 3:
	tmp = 2*PI*ajdi*x[j];
	vx[i*ns+j] = -tmp;
	if(sjl != 0)
	  Sx[m2v[i*nr+i]*ns+j] = 2*PI*x[j]*tmp/sjl;
	
	tmp = 2*PI*ajdi*y[j];
	vy[i*ns+j] = -tmp;
	if(sjl != 0)
	  Sy[m2v[i*nr+i]*ns+j] = 2*PI*y[j]*tmp/sjl;
	
	tmp = 2*PI*ajdi*z[j];
	vz[i*ns+j] = -tmp;
	if(sjl != 0)
	  Sz[m2v[i*nr+i]*ns+j] = 2*PI*z[j]*tmp/sjl;
	break;
      }      
    }
    
    /* non-separable term */
    k = j;
    for(i=0;i<nr;i++){
      for(j=0;j<i;j++){  
	ajai_r = -Ar[i*ns+k]*Ar[j*ns+k] - Ai[i*ns+k]*Ai[j*ns+k];
	ajai_i = -Ar[i*ns+k]*Ai[j*ns+k] + Ai[i*ns+k]*Ar[j*ns+k];
	
	/* calculate phase angle */
	sji = atan2(ajai_i,ajai_r);
	
	switch(nd){
	case 1:
	  tmp = 2*PI*x[k]*ajai_i;
	  vx[i*ns+k] += tmp;
	  vx[j*ns+k] -= tmp;
	  
	  if(sji != 0)
	    {
	      tmp2 = 2*PI*x[k]*tmp/sji;
	      Sx[m2v[i*nr+i]*ns+k] += tmp2;
	      Sx[m2v[j*nr+i]*ns+k] = -tmp2;
	      Sx[m2v[j*nr+j]*ns+k] += tmp2;
	      /*Sx[m2v[i*nr+j]*ns+k] = -tmp2;*/
	    }
	  break;
	case 2:
	  tmp = 2*PI*x[k]*ajai_i;
	  vx[i*ns+k] += tmp;
	  vx[j*ns+k] -= tmp;
	  
	  if(sji != 0)
	    {
	      tmp2 = 2*PI*x[k]*tmp/sji;
	      Sx[m2v[i*nr+i]*ns+k] += tmp2;
	      Sx[m2v[j*nr+i]*ns+k] = -tmp2;
	      Sx[m2v[j*nr+j]*ns+k] += tmp2;
	      /*Sx[m2v[i*nr+j]*ns+k] = -tmp2;*/
	    }
	  
	  tmp = 2*PI*y[k]*ajai_i;
	  vy[i*ns+k] += tmp;
	  vy[j*ns+k] -= tmp;
	  
	  if(sji != 0)
	    {
	      tmp2 = 2*PI*y[k]*tmp/sji;
	      Sy[m2v[i*nr+i]*ns+k] += tmp2;
	      Sy[m2v[j*nr+i]*ns+k] = -tmp2;
	      Sy[m2v[j*nr+j]*ns+k] += tmp2;
	      /*Sy[m2v[i*nr+j]*ns+k] = -tmp2;*/
	    }
	  break;
	case 3:
	  tmp = 2*PI*x[k]*ajai_i;
	  vx[i*ns+k] += tmp;
	  vx[j*ns+k] -= tmp;
	  
	  if(sji != 0){
	    tmp2 = 2*PI*x[k]*tmp/sji;
	    Sx[m2v[i*nr+i]*ns+k] += tmp2;
	    Sx[m2v[j*nr+i]*ns+k] = -tmp2;
	    Sx[m2v[j*nr+j]*ns+k] += tmp2;
	    /*Sx[m2v[i*nr+j]*ns+k] -= tmp2;*/
	  }
	  
	  tmp = 2*PI*y[k]*ajai_i;
	  vy[i*ns+k] += tmp;
	  vy[j*ns+k] -= tmp;
	  
	  if(sji != 0){
	    tmp2 = 2*PI*y[k]*tmp/sji;
	    Sy[m2v[i*nr+i]*ns+k] += tmp2;
	    Sy[m2v[j*nr+i]*ns+k] = -tmp2;
	    Sy[m2v[j*nr+j]*ns+k] += tmp2;
	    /*Sy[m2v[i*nr+j]*ns+k] -= tmp2;*/
	  }
	  
	  tmp = 2*PI*z[k]*ajai_i;
	  vz[i*ns+k] += tmp;
	  vz[j*ns+k] -= tmp;
	  
	  if(sji != 0){
	    tmp2 = 2*PI*z[k]*tmp/sji;
	    Sz[m2v[i*nr+i]*ns+k] += tmp2;
	    Sz[m2v[j*nr+i]*ns+k] = -tmp2;
	    Sz[m2v[j*nr+j]*ns+k] += tmp2;
	    /*Sz[m2v[i*nr+j]*ns+k] -= tmp2;*/
	  }  
	  break;
	} /* nd switch */
	
      } /* nr inner loop */
    } /* nr outer loop */
    
    j = k;
    j += blockDim.x * gridDim.x;
    
  } /* while j < ns */

}


